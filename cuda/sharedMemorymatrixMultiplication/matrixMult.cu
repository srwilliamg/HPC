#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <fstream>

using namespace std;

#define TILE_WIDTH 16

__global__ void MatrixMulKernel(float *d_M, float *d_N, float *d_P,int width){
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	
	int row = by * TILE_WIDTH + ty;
	int col = bx * TILE_WIDTH + tx;

	float Pvalue = 0;

	for (int i = 0; i < width/TILE_WIDTH; ++i){

		Mds[ty][tx] = d_M[row*width + i*TILE_WIDTH + tx];
		Nds[ty][tx] =  d_N[(i*TILE_WIDTH + ty) *width + col];
		__syncthreads();

		for (int j = 0; j < TILE_WIDTH; ++j){
			Pvalue += Mds[ty][j] * Nds[j][tx];
		}
		__syncthreads();
	}
	d_P[row*width + col] = Pvalue;
}

int matrixMulHost(float *h_M, float *h_N, float *h_P, int width){
    int Pvalue;

    for(int row = 0; row < width ; ++row){
        for(int col = 0; col < width ; ++col){
            Pvalue = 0;
            for(int k = 0; k < width ; ++k){
                Pvalue += h_M[row*width+k] * h_N[k*width+col];
            }
            h_P[row*width+col] = Pvalue;
        }
    }
    return 0;
}

int initValues(float *data, int width){
    for(int i = 0; i < width*width; i++)
        data[i] = 2;
    return 0;
}

int printData(float *data, int width){
    for(int i = 0; i < width; ++i){
        for(int j = 0; j < width; ++j){
            printf("%f ", data[(i*width)+j]);
        }
        printf("\n");
    }
    return 0;
}

int main(int argc, char const *argv[])
{
	float *h_M, *h_N, *h_P,*h_P_d;
    float *d_M, *d_N,*d_P;
    std::string num = argv[1];
    int width = std::stoi(num);
    int size = width * width * sizeof(float);
    clock_t start, end, startGPU, endGPU;
    double cpu_time_used, gpu_time_used, aceleration;

   //for (int times = 0; times < 20; times++){
        h_M = (float*)malloc(size);
        h_N = (float*)malloc(size);
        h_P = (float*)malloc(size);
        h_P_d = (float*)malloc(size);

        initValues(h_M, width);
        initValues(h_N, width);

        printData(h_M, width);

        /////////Algoritmo Secuencial////////////////////////////////////////////
        start = clock();
        matrixMulHost(h_M, h_N, h_P, width);
        end = clock();
        cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
        printf("Tiempo algoritmo secuencial: %.10f\n", cpu_time_used);
        /////////Algoritmo Secuencial/////////////////////////////////////////////

        hipMalloc((void**)&d_M,size);
        hipMalloc((void**)&d_N,size);
        hipMalloc((void**)&d_P,size);
        //////////////////////Algoritmo Paralelo///////////////////////////
        startGPU = clock();
        hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
        hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

        int blockSize = 32;
        dim3 dimBlock(blockSize,blockSize,1);
        dim3 dimGrid(ceil(width/float(blockSize)),ceil(width/float(blockSize)),1);
        MatrixMulKernel<<<dimGrid,dimBlock>>>(d_M,d_N,d_P,width);
        hipDeviceSynchronize();
        hipMemcpy(h_P_d,d_P,size,hipMemcpyDeviceToHost);
        endGPU = clock();
        gpu_time_used = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
        aceleration = cpu_time_used/gpu_time_used;

	    printf("%s \n","here i am");
        printData(h_P_d,width);

        printf("Tiempo algoritmo paralelo: %.10f\n", gpu_time_used);
        printf("La aceleración obtenida es de %.10fX\n",aceleration);

        std::string name =  "TimesMult.txt"+num;

        ofstream outfile(name,ios::binary | ios::app);
        outfile << gpu_time_used<<" "<< cpu_time_used <<" "<< aceleration << "\n";
        outfile.close();

        free(h_M);
        free(h_N);
        free(h_P);
        free(h_P_d);
        hipFree(d_M);
        hipFree(d_N);
        hipFree(d_P);
    //}

	return 0;
}
